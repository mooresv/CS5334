#include<hip/hip_runtime.h>
#include <stdio.h>

int main()
{
   hipDeviceProp_t Props;
   hipGetDeviceProperties( &Props,0);

   printf("shared mem: %d)\n", Props.sharedMemPerBlock);
   printf("max threads/block: %d\n",Props.maxThreadsPerBlock);
   printf("max blocks: %d\n",Props.maxGridSize[0]);
   printf("total Const mem: %d\n",Props.totalConstMem);
}
