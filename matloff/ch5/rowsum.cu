#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// CUDA example:  finds row sums of an integer matrix m

// find1elt() finds the rowsum of one row of the nxn matrix m, storing the
// result in the corresponding position in the rowsum array rs; matrix
// stored as 1-dimensional, row-major order

__global__ void find1elt(int *m, int *rs, int n)
{
   int rownum = blockIdx.x;  // this thread will handle row # rownum
   int sum = 0;
   for (int k = 0; k < n; k++)
      sum += m[rownum*n+k];
   rs[rownum] = sum;
}

int main(int argc, char **argv)
{
    int n = atoi(argv[1]);  // number of matrix rows/cols
    int *hm, // host matrix
        *dm, // device matrix
        *hrs, // host rowsums
        *drs; // device rowsums
    int msize = n * n * sizeof(int);  // size of matrix in bytes
    // allocate space for host matrix
    hm = (int *) malloc(msize);  
    // as a test, fill matrix with consecutive integers
    int t = 0,i,j;
    for (i = 0; i < n; i++) {
       for (j = 0; j < n; j++) {
          hm[i*n+j] = t++;
       }
    }
    // allocate space for device matrix 
    hipMalloc((void **)&dm,msize);
    // copy host matrix to device matrix
    hipMemcpy(dm,hm,msize,hipMemcpyHostToDevice);
    // allocate host, device rowsum arrays
    int rssize = n * sizeof(int);
    hrs = (int *) malloc(rssize);  
    hipMalloc((void **)&drs,rssize);
    // set up parameters for threads structure
    dim3 dimGrid(n,1);  // n blocks 
    dim3 dimBlock(1,1,1);  // 1 thread per block
    // invoke the kernel
    find1elt<<<dimGrid,dimBlock>>>(dm,drs,n);
    // wait for kernel to finish
    hipDeviceSynchronize();
    // copy row vector from device to host
    hipMemcpy(hrs,drs,rssize,hipMemcpyDeviceToHost);
    // check results
    if (n < 10) for(int i=0; i<n; i++) printf("%d\n",hrs[i]);
    // clean up
    free(hm);
    hipFree(dm);
    free(hrs);
    hipFree(drs);
}
